// Example 2. Application Using C and cuBLAS: 0-based indexing
//-----------------------------------------------------------
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#include "hipblaslt.h"
#include "lib/utils.cuh"

#define SIZES_START 2048
#define SIZES_END 2048
#define TIMES 10000
#define WARMUP 5000

int main(void) {
    int i, j, N;

    FILE* fptr;
    fptr = fopen("timings.txt", "w");
    if (fptr == NULL) {
        printf("Error!");
        exit(1);
    }

    // For loop over sizes
    for (N = SIZES_START; N <= SIZES_END; N++) {
        float* a = (float*)malloc(N * N * sizeof(float));
        float* b = (float*)malloc(N * N * sizeof(float));
        float* c = (float*)malloc(N * N * sizeof(float));

        // initialize matrix a
        for (j = 0; j < N; j++) {
            for (i = 0; i < N; i++) {
                a[IDX2C(i, j, N)] = ((float)(i * N + j + 1)) / ((float)(N * N));
            }
        }

        float* A;
        checkCudaStatus(hipMalloc((void**)&A, N * N * sizeof(*a)));
        float* B;
        checkCudaStatus(hipMalloc((void**)&B, N * N * sizeof(*b)));
        float* C;
        checkCudaStatus(hipMalloc((void**)&C, N * N * sizeof(*c)));

        // create cublasLt handle
        hipblasLtHandle_t handle;
        checkCublasStatus(hipblasLtCreate(&handle));

        // create operation desciriptor
        hipblasLtMatmulDesc_t operationDesc;
        checkCublasStatus(hipblasLtMatmulDescCreate(&operationDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F));
        hipblasOperation_t transa = HIPBLAS_OP_N;
        checkCublasStatus(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(transa)));
        hipblasOperation_t transb = HIPBLAS_OP_T;
        checkCublasStatus(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transb)));

        checkCublasStatus(hipblasSetMatrix(N, N, sizeof(*a), a, N, A, N));

        // create (empty) preference for heuristics
        hipblasLtMatmulPreference_t preference;
        checkCublasStatus(hipblasLtMatmulPreferenceCreate(&preference));

        // create heuristic
        int returnedResults = 0;
        hipblasLtMatmulHeuristicResult_t heuristicResult = {};
        hipblasLtMatrixLayout_t Adesc = NULL, Bdesc = NULL, Cdesc = NULL;
        // create matrix descriptors, we are good with the details here so no need to set any extra attributes
        checkCublasStatus(hipblasLtMatrixLayoutCreate(&Adesc, HIP_R_32F, N, N, N));
        checkCublasStatus(hipblasLtMatrixLayoutCreate(&Bdesc, HIP_R_32F, N, N, N));
        checkCublasStatus(hipblasLtMatrixLayoutCreate(&Cdesc, HIP_R_32F, N, N, N));

        checkCublasStatus(hipblasLtMatmulAlgoGetHeuristic(handle, operationDesc, Adesc, Bdesc, Cdesc, Cdesc, preference, 1, &heuristicResult, &returnedResults));
        if (returnedResults == 0) {
            checkCublasStatus(HIPBLAS_STATUS_NOT_SUPPORTED);
        }

        hipEvent_t start, end;
        hipEventCreate(&start);
        hipEventCreate(&end);

        // Perform GEMM AA^T + 0
        float alpha = 1.0f;
        float beta = 0.0f;
        for (int i = 0; i < WARMUP; i++) {
            checkCublasStatus(
                hipblasLtMatmul(
                    handle,
                    operationDesc,
                    &alpha,
                    A,
                    Adesc,
                    B,
                    Bdesc,
                    &beta,
                    C,
                    Cdesc,
                    C,
                    Cdesc,
                    &heuristicResult.algo,
                    nullptr,
                    0,
                    0));
        }
        hipEventRecord(start);
        for (int i = 0; i < TIMES; i++) {
            checkCublasStatus(
                hipblasLtMatmul(
                    handle,
                    operationDesc,
                    &alpha,
                    A,
                    Adesc,
                    B,
                    Bdesc,
                    &beta,
                    C,
                    Cdesc,
                    C,
                    Cdesc,
                    &heuristicResult.algo,
                    nullptr,
                    0,
                    0));
        }
        hipEventRecord(end);
        hipEventSynchronize(end);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, end);

        printf("N = %d, %.4f ops/ms\n", N, TIMES / milliseconds);
        fprintf(fptr, "%d,%f\n", N, TIMES / milliseconds);
        fflush(fptr);

        // Deallocate
        checkCublasStatus(hipblasLtMatmulPreferenceDestroy(preference));
        checkCublasStatus(hipblasLtMatmulDescDestroy(operationDesc));
        checkCublasStatus(hipblasLtDestroy(handle));

        checkCudaStatus(hipFree(A));
        checkCudaStatus(hipFree(B));
        checkCudaStatus(hipFree(C));
        free(a);
        free(b);
        free(c);
    }

    fclose(fptr);
    return EXIT_SUCCESS;
}