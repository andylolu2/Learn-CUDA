#include <stdio.h>

#include "lib/utils.cuh"

int main(void) {
    // allocate host value
    float n = 1234.56f;
    float *a_host = &n;

    // allocate device memory
    float *a_device, *b_device;
    checkCudaStatus(hipMalloc(&a_device, sizeof(float)));
    checkCudaStatus(hipMalloc(&b_device, sizeof(float)));

    // shuffle things around
    checkCudaStatus(hipMemcpy(a_device, a_host, sizeof(float), hipMemcpyHostToDevice));
    checkCudaStatus(hipMemcpy(b_device, a_device, sizeof(float), hipMemcpyDeviceToDevice));
    checkCudaStatus(hipMemcpy(a_host, b_device, sizeof(float), hipMemcpyDeviceToHost));

    // free memory
    checkCudaStatus(hipFree(a_device));
    checkCudaStatus(hipFree(b_device));

    printf("%f\n", *a_host);

    return 0;
}
