#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdio.h>

#include <iostream>

#include "lib/utils.cuh"
#define N (1 << 10)

__global__ void kernel(float* array) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    array[idx] = 1000 * blockIdx.x + threadIdx.x;
}

int main(void) {
    // allocate device memory
    float* a_device;
    checkCudaStatus(hipMalloc(&a_device, N * sizeof(float)));

    kernel<<<1, N>>>(a_device);

    // allocate host memory
    float* a_host = (float*)malloc(N * sizeof(float));

    // do the work
    checkCudaStatus(hipMemcpy(a_host, a_device, N * sizeof(float), hipMemcpyDeviceToHost));

    // deallocate
    checkCudaStatus(hipFree(a_device));

    return 0;
}
