#include <cudnn_frontend.h>
#include <cudnn_frontend_find_plan.h>
#include <cudnn_frontend_get_plan.h>

#include <array>
#include <iostream>

#include "lib/utils.cuh"

// Method for engine config generator based on heuristics
auto heurgen_method = [](cudnn_frontend::OperationGraph &opGraph) -> cudnn_frontend::EngineConfigList {
    auto heuristics = cudnn_frontend::EngineHeuristicsBuilder()
                          .setOperationGraph(opGraph)
                          .setHeurMode(CUDNN_HEUR_MODE_A)
                          .build();
    std::cout << "Heuristic has " << heuristics.getEngineConfigCount() << " configurations " << std::endl;

    auto &engine_configs = heuristics.getEngineConfig(heuristics.getEngineConfigCount());
    return engine_configs;
};

// Method for engine config generator based on fallback list
auto fallback_method = [](cudnn_frontend::OperationGraph &opGraph) -> cudnn_frontend::EngineConfigList {
    auto fallback = cudnn_frontend::EngineFallbackListBuilder()
                        .setOperationGraph(opGraph)
                        // .setOperation(CUDNN_BACKEND_OPERATION_CONVOLUTION_FORWARD_DESCRIPTOR)
                        .build();
    auto &fallback_list = fallback.getFallbackList();
    return fallback_list;
};

int main(int argc, char *argv[]) {
    if (argc != 3) {
        printf("Usage: %s N TIMES\n", argv[0]);
        return 0;
    }

    int N = atoi(argv[1]);
    int TIMES = atoi(argv[2]);
    printf("N = %d, TIMES = %d\n", N, TIMES);

    printf("CUDNN VERSION FROM hipdnnGetVersion(): %zu\n", hipdnnGetVersion());
    hipdnnHandle_t handle;
    checkCudnnErr(hipdnnCreate(&handle));

    int64_t dims[3] = {1, N, N};
    int64_t stride[3] = {N * N, N, 1};

    auto xTensor = cudnn_frontend::TensorBuilder()
                       .setDim(3, dims)
                       .setStride(3, stride)
                       .setId('x')
                       .setAlignment(16)  // 16B alignment is needed to run a tensor core engine
                       .setDataType(HIPDNN_DATA_HALF)
                       .build();

    auto yTensor = cudnn_frontend::TensorBuilder()
                       .setDim(3, dims)
                       .setStride(3, stride)
                       .setId('y')
                       .setAlignment(16)
                       .setDataType(HIPDNN_DATA_HALF)
                       .build();

    auto cTensor = cudnn_frontend::TensorBuilder()
                       .setDim(3, dims)
                       .setStride(3, stride)
                       .setId('c')
                       .setAlignment(16)
                       .setDataType(HIPDNN_DATA_HALF)
                       .build();

    auto matmulDesc = cudnn_frontend::MatMulDescBuilder()
                          .setComputeType(HIPDNN_DATA_HALF)
                          .build();

    auto matmulOp = cudnn_frontend::OperationBuilder(CUDNN_BACKEND_OPERATION_MATMUL_DESCRIPTOR)
                        .setaMatDesc(xTensor)
                        .setbMatDesc(yTensor)
                        .setcMatDesc(cTensor)
                        .setmatmulDesc(matmulDesc)
                        .build();

    std::array<cudnn_frontend::Operation const *, 1> ops = {&matmulOp};

    auto opGraph = cudnn_frontend::OperationGraphBuilder()
                       .setHandle(handle)
                       .setOperationGraph(ops.size(), ops.data())
                       .build();

    size_t xSize = N * N * sizeof(half);
    size_t ySize = N * N * sizeof(half);
    size_t cSize = N * N * sizeof(half);
    printf("xSize = %zu, ySize = %zu, cSize = %zu\n", xSize, ySize, cSize);
    void *x_ptr, *y_ptr, *c_ptr;
    checkCudaStatus(hipMalloc(&x_ptr, xSize));
    checkCudaStatus(hipMalloc(&y_ptr, ySize));
    checkCudaStatus(hipMalloc(&c_ptr, cSize));

    void *data_ptrs[] = {x_ptr, y_ptr, c_ptr};
    int64_t uids[] = {'x', 'y', 'c'};

    auto variantPack = cudnn_frontend::VariantPackBuilder()
                           .setDataPointers(3, data_ptrs)
                           .setUids(3, uids)
                           .build();
    std::cout << "variantPack " << variantPack.describe() << std::endl;

    std::array<cudnn_frontend::GeneratorSource const, 2> sources = {heurgen_method, fallback_method};
    cudnn_frontend::EngineConfigGenerator generator(static_cast<int>(sources.size()), sources.data());

    auto options = generator.cudnnFindPlan<cudnn_frontend::CudnnFindSamplingTechnique::CUDNN_FIND_SAMPLE_TILL_STABLE>(
        handle, opGraph, variantPack);
    cudnn_frontend::ExecutionPlan plan = options.front();
    std::cout << "Plan chosen: " << plan.getTag() << std::endl;

    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    for (int i = 0; i < 100; i++) {
        checkCudnnErr(
            cudnnBackendExecute(
                handle, plan.get_raw_desc(), variantPack.get_raw_desc()));
    }

    hipEventRecord(start);
    for (int i = 0; i < TIMES; i++) {
        checkCudnnErr(
            cudnnBackendExecute(
                handle, plan.get_raw_desc(), variantPack.get_raw_desc()));
    }
    hipEventRecord(end);
    hipEventSynchronize(end);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, end);
    double nOps = 2 * (double)TIMES * (double)N * (double)N * (double)N / ((double)milliseconds / 1000.0);
    printf("N = %d, %.4f ops/ms, %.4f TFLOPS\n", N, TIMES / milliseconds, nOps / 1e12);

    checkCudaStatus(hipFree(x_ptr));
    checkCudaStatus(hipFree(y_ptr));
    checkCudaStatus(hipFree(c_ptr));
    checkCudnnErr(hipdnnDestroy(handle));
    return 0;
}
