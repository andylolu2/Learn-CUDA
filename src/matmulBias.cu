// Example 2. Application Using C and cuBLAS: 0-based indexing
//-----------------------------------------------------------
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#include <iomanip>
#include <iostream>

#include "hipblaslt.h"
#include "lib/utils.cuh"

int main(void) {
    int M = 16;
    int N = 16;
    int K = 16;

    float* a = (float*)malloc(K * M * sizeof(float));
    float* b = (float*)malloc(K * N * sizeof(float));
    float* c = (float*)malloc(N * M * sizeof(float));
    float* bias = (float*)malloc(N * sizeof(float));

    // initialize matrix a and b
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < K; j++) {
            a[i * K + j] = static_cast<float>(i * K + j);  // / static_cast<float>(M * K);
        }
    }
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < K; j++) {
            b[i * K + j] = static_cast<float>(i * K + j);  // / static_cast<float>(N * K);
        }
    }
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            c[i * K + j] = 0;
        }
    }
    for (int i = 0; i < N; i++) {
        bias[i] = static_cast<float>(i) / 10;  // / static_cast<float>(N * K);
    }

    float* A;
    float* B;
    float* C;
    float* Bias;
    checkCudaStatus(hipMalloc(&A, M * K * sizeof(float)));
    checkCudaStatus(hipMalloc(&B, N * K * sizeof(float)));
    checkCudaStatus(hipMalloc(&C, M * N * sizeof(float)));
    checkCudaStatus(hipMalloc(&Bias, N * sizeof(float)));
    hipMemcpy(A, a, M * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(B, b, N * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(C, c, M * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(Bias, bias, N * sizeof(float), hipMemcpyHostToDevice);

    // create cublasLt handle
    hipblasLtHandle_t handle;
    checkCublasStatus(hipblasLtCreate(&handle));

    // create operation desciriptor
    hipblasLtMatmulDesc_t operationDesc;
    checkCublasStatus(hipblasLtMatmulDescCreate(&operationDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F));
    hipblasOperation_t transa = HIPBLAS_OP_N;
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(transa)));
    hipblasOperation_t transb = HIPBLAS_OP_N;
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transb)));

    // create (empty) preference for heuristics
    hipblasLtMatmulPreference_t preference;
    checkCublasStatus(hipblasLtMatmulPreferenceCreate(&preference));

    // create heuristic
    int returnedResults = 0;
    hipblasLtMatmulHeuristicResult_t heuristicResult;
    hipblasLtMatrixLayout_t Adesc;
    hipblasLtMatrixLayout_t Bdesc;
    hipblasLtMatrixLayout_t Biasdesc;
    hipblasLtMatrixLayout_t Cdesc;
    // create matrix descriptors, we are good with the details here so no need to set any extra attributes
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&Adesc, HIP_R_32F, M, K, M));
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&Bdesc, HIP_R_32F, N, K, N));
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&Biasdesc, HIP_R_32F, M, N, 0));
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&Cdesc, HIP_R_32F, M, N, M));

    checkCublasStatus(hipblasLtMatmulAlgoGetHeuristic(handle, operationDesc, Adesc, Bdesc, Biasdesc, Cdesc, preference, 1, &heuristicResult, &returnedResults));
    if (returnedResults == 0) {
        checkCublasStatus(HIPBLAS_STATUS_NOT_SUPPORTED);
    }

    float alpha = 1.0f;
    float beta = 1.0f;
    checkCublasStatus(hipblasLtMatmul(
        handle,
        operationDesc,
        &alpha,
        A,
        Adesc,
        B,
        Bdesc,
        &beta,
        Bias,
        Biasdesc,
        C,
        Cdesc,
        &heuristicResult.algo,
        nullptr,
        0,
        0));

    hipMemcpy(c, C, N * M * sizeof(float), hipMemcpyDeviceToHost);

    std::cout << "A:" << std::endl;
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            std::cout << a[i * N + j] << " ";
        }
        std::cout << std::endl;
    }
    std::cout << "B:" << std::endl;
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            std::cout << b[i * N + j] << " ";
        }
        std::cout << std::endl;
    }
    std::cout << "bias:" << std::endl;
    for (int i = 0; i < N; i++) {
        std::cout << bias[i] << " ";
    }
    std::cout << std::endl;
    std::cout << "C:" << std::endl;
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            std::cout << std::setprecision(8) << c[i * N + j] << " ";
        }
        std::cout << std::endl;
    }

    // Deallocate
    checkCublasStatus(hipblasLtMatmulPreferenceDestroy(preference));
    checkCublasStatus(hipblasLtMatmulDescDestroy(operationDesc));
    checkCublasStatus(hipblasLtDestroy(handle));

    checkCudaStatus(hipFree(A));
    checkCudaStatus(hipFree(B));
    checkCudaStatus(hipFree(C));
    free(a);
    free(b);
    free(c);

    return 0;
}